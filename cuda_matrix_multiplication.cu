#include "hip/hip_runtime.h"
/*
  Matrix Multiplication w/GPU (cuda)
  Víctor Rendón Suárez
  A01022462
*/
#include <hip/hip_runtime.h>
#include <chrono>
#include "common.h"

using namespace std;
#define SIZE 1000

void initialize_matrix(int *matrix, int n)
{
  for (int i = 0; i < n * n; i++)
    matrix[i] = i;
}

__global__ void multiply_matrix_cuda(int *matrixA, int *matrixB, long *result, int n)
{
  int ix = threadIdx.x + blockIdx.x * blockDim.x;
  int iy = blockIdx.y;
  if(ix < n && iy < n) {
    long add = 0;
    for (int i = 0; i < n; i++) {
      add += matrixA[iy * n + i] * matrixB[i * n + ix];
    }
    result[iy * n + ix] = add;
  }
}

int main(int argc, char const *argv[])
{
  // Setup device
  int dev = 0;
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, dev);
  printf("Using Device %d: %s\n", dev, deviceProp.name);
  hipSetDevice(dev);

  // Specify size
  int n = SIZE;
  int bytes = n * n * sizeof(int);
  int lngBytes = n * n * sizeof(long);
  // Matrix definition
  int *matrixA = (int *) malloc(bytes);
  int *matrixB = (int *) malloc(bytes);
  long *result = (long *) malloc(lngBytes);
  int *d_matrixA;
  int *d_matrixB;
  long *d_result_matrix;

  // Initialize matrices
  initialize_matrix(matrixA, n);
  initialize_matrix(matrixB, n);

  // Allocate device memory
  hipMalloc((void **)&d_matrixA, bytes);
  hipMalloc((void **)&d_matrixB, bytes);
  hipMalloc((void **)&d_result_matrix, lngBytes);

  // Transfer data from host to device
  hipMemcpy(d_matrixA, matrixA, bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_matrixB, matrixB, bytes, hipMemcpyHostToDevice);

  // Kernel configuration
  int dimx = 32;
  dim3 block(dimx, 1);
  dim3 grid((n + block.x - 1) / block.x, n);

  // Multiply the matrices using GPU, measure elapsed time
  auto start_time = chrono::high_resolution_clock::now();
  multiply_matrix_cuda<<<grid, block>>>(d_matrixA, d_matrixB, d_result_matrix, n);
  hipDeviceSynchronize();
  auto end_time = chrono::high_resolution_clock::now();
  chrono::duration<float, std::milli> duration_ms = end_time - start_time;
  printf("Matrix multiplication on GPU, time elapsed: %f ms\n", duration_ms.count());

  // Copy result to host
  hipMemcpy(result, d_result_matrix, lngBytes, hipMemcpyDeviceToHost);

  // Free allocated memory
  hipFree(d_matrixA);
  hipFree(d_matrixB);
  hipFree(d_result_matrix);
  free(matrixA);
  free(matrixB);
  free(result);

  hipDeviceReset();

  return 0;
}
